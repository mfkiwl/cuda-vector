#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include "utility.cuh"
using namespace std;

#define BSIZE 1024
#define PROB 90

// Low Level API
class VectorMemMap {
private:
    hipDeviceptr_t d_p;
    hipMemAllocationProp prop;
    hipMemAccessDesc accessDesc;
    struct Range {
        hipDeviceptr_t start;
        size_t sz;
    };
    std::vector<Range> va_ranges;
    std::vector<hipMemGenericAllocationHandle_t> handles;
    std::vector<size_t> handle_sizes;
    size_t alloc_sz;
    size_t reserve_sz;
    size_t chunk_sz;
public:
    VectorMemMap(hipCtx_t context);
    ~VectorMemMap();

    hipDeviceptr_t getPointer() const {
        return d_p;
    }

    size_t getSize() const {
        return alloc_sz;
    }
    size_t getReserve() const {
        return reserve_sz;
    }

    // Reserves some extra space in order to speed up grow()
    hipError_t reserve(size_t new_sz);

    // Actually commits num bytes of additional memory
    hipError_t grow(size_t new_sz);
};

VectorMemMap::VectorMemMap(hipCtx_t context) : d_p(0ULL), prop(), handles(), alloc_sz(0ULL), reserve_sz(0ULL), chunk_sz(0ULL)
{
    hipDevice_t device;
    hipCtx_t prev_ctx;
    hipError_t status = hipSuccess;
    (void)status;

    status = hipCtxGetCurrent(&prev_ctx);
    assert(status == hipSuccess);
    if (hipCtxSetCurrent(context) == hipSuccess) {
        status = hipCtxGetDevice(&device);
        assert(status == hipSuccess);
        status = hipCtxSetCurrent(prev_ctx);
        assert(status == hipSuccess);
    }

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = (int)device;
    prop.win32HandleMetaData = NULL;

    accessDesc.location = prop.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    status = hipMemGetAllocationGranularity(&chunk_sz, &prop, hipMemAllocationGranularityMinimum);
    assert(status == hipSuccess);
}

VectorMemMap::~VectorMemMap()
{
    hipError_t status = hipSuccess;
    (void)status;
    if (d_p != 0ULL) {
        status = hipMemUnmap(d_p, alloc_sz);
        assert(status == hipSuccess);
        for (size_t i = 0ULL; i < va_ranges.size(); i++) {
            status = hipMemAddressFree(va_ranges[i].start, va_ranges[i].sz);
            assert(status == hipSuccess);
        }
        for (size_t i = 0ULL; i < handles.size(); i++) {
            status = hipMemRelease(handles[i]);
            assert(status == hipSuccess);
        }
    }
}

hipError_t
VectorMemMap::reserve(size_t new_sz)
{
    hipError_t status = hipSuccess;
    hipDeviceptr_t new_ptr = 0ULL;

    if (new_sz <= reserve_sz) {
        return hipSuccess;
    }

    const size_t aligned_sz = ((new_sz + chunk_sz - 1) / chunk_sz) * chunk_sz;

    status = hipMemAddressReserve(&new_ptr, (aligned_sz - reserve_sz), 0ULL, d_p + reserve_sz, 0ULL);

    // Try to reserve an address just after what we already have reserved
    if (status != hipSuccess || (new_ptr != d_p + reserve_sz)) {
        if (new_ptr != 0ULL) {
            (void)hipMemAddressFree(new_ptr, (aligned_sz - reserve_sz));
        }
        // Slow path - try to find a new address reservation big enough for us
        status = hipMemAddressReserve(&new_ptr, aligned_sz, 0ULL, 0U, 0);
        if (status == hipSuccess && d_p != 0ULL) {
            hipDeviceptr_t ptr = new_ptr;
            // Found one, now unmap our previous allocations
            status = hipMemUnmap(d_p, alloc_sz);
            assert(status == hipSuccess);
            for (size_t i = 0ULL; i < handles.size(); i++) {
                const size_t hdl_sz = handle_sizes[i];
                // And remap them, enabling their access
                if ((status = hipMemMap(ptr, hdl_sz, 0ULL, handles[i], 0ULL)) != hipSuccess)
                    break;
                if ((status = hipMemSetAccess(ptr, hdl_sz, &accessDesc, 1ULL)) != hipSuccess)
                    break;
                ptr += hdl_sz;
            }
            if (status != hipSuccess) {
                // Failed the mapping somehow... clean up!
                status = hipMemUnmap(new_ptr, aligned_sz);
                assert(status == hipSuccess);
                status = hipMemAddressFree(new_ptr, aligned_sz);
                assert(status == hipSuccess);
            }
            else {
                // Clean up our old VA reservations!
                for (size_t i = 0ULL; i < va_ranges.size(); i++) {
                    (void)hipMemAddressFree(va_ranges[i].start, va_ranges[i].sz);
                }
                va_ranges.clear();
            }
        }
        // Assuming everything went well, update everything
        if (status == hipSuccess) {
            Range r;
            d_p = new_ptr;
            reserve_sz = aligned_sz;
            r.start = new_ptr;
            r.sz = aligned_sz;
            va_ranges.push_back(r);
        }
    }
    else {
        Range r;
        r.start = new_ptr;
        r.sz = aligned_sz - reserve_sz;
        va_ranges.push_back(r);
        if (d_p == 0ULL) {
            d_p = new_ptr;
        }
        reserve_sz = aligned_sz;
    }

    return status;
}

hipError_t
VectorMemMap::grow(size_t new_sz)
{
    hipError_t status = hipSuccess;
    hipMemGenericAllocationHandle_t handle;
    if (new_sz <= alloc_sz) {
        return hipSuccess;
    }

    const size_t size_diff = new_sz - alloc_sz;
    // Round up to the next chunk size
    const size_t sz = ((size_diff + chunk_sz - 1) / chunk_sz) * chunk_sz;

    if ((status = reserve(alloc_sz + sz)) != hipSuccess) {
        return status;
    }

    if ((status = hipMemCreate(&handle, sz, &prop, 0)) == hipSuccess) {
        if ((status = hipMemMap(d_p + alloc_sz, sz, 0ULL, handle, 0ULL)) == hipSuccess) {
            if ((status = hipMemSetAccess(d_p + alloc_sz, sz, &accessDesc, 1ULL)) == hipSuccess) {
                handles.push_back(handle);
                handle_sizes.push_back(sz);
                alloc_sz += sz;
            }
            if (status != hipSuccess) {
                (void)hipMemUnmap(d_p + alloc_sz, sz);
            }
        }
        if (status != hipSuccess) {
            (void)hipMemRelease(handle);
        }
    }

    return status;
}

__device__ int &at(hipDeviceptr_t d_p, unsigned int i) {
	return (*(int**)&d_p)[i];
}

__device__ void insert_atomic(hipDeviceptr_t d_p, int e, int *size, int q) {
	int idx = atomicAdd(size, 1);
	at(d_p, idx) = e;
}

__global__ void initVec(hipDeviceptr_t d_p, unsigned int n, int* in) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n) return;
	at(d_p, tid) = in[tid];

}

__global__ void test(hipDeviceptr_t d_p, size_t n) {
	for (size_t i = 0; i < n; ++i) {
		at(d_p, i) = i;
	}
}

__global__ void printVec(hipDeviceptr_t d_p, size_t n) {
	for (size_t i = 0; i < n; ++i) {
		printf("%d ", at(d_p, i));
	}
	printf("\n");
}

__global__ void test_insert(hipDeviceptr_t d_p, int *size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= *size) return;
	int idx = atomicAdd(size, 1);
	at(d_p, idx) = tid;
}

// low level api test
void run_experiment(hipCtx_t ctx) {
	int rep = 10;
	int size = 1024*100;
	int *ds;
	hipMalloc(&ds, sizeof(int));
	hipMemcpy(ds, &size, sizeof(int), hipMemcpyHostToDevice);

	float results[rep];
	float s = 0.0;

	VectorMemMap a = VectorMemMap(ctx);
	hipError_t status;

	status = a.grow(size*sizeof(int));
	test<<<1,1>>>(a.getPointer(), size); kernelCallCheck();
	
	for (int i = 0; i < rep; ++i) {
		hipEvent_t start, stop;
		start_clock(start, stop);
		status = a.grow(size*2*sizeof(int));
		test_insert<<<gridSize(size, 1024), 1024>>>(a.getPointer(), ds);
		results[i] = stop_clock(start, stop);
		s += results[i];
		size *= 2;
	}

	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results[i]);
	}
	printf("%f\n", results[rep-1]);
	printf("%f\n", s);
}


// static
__global__ void initVec(int *array, unsigned int n, int* in) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n) return;
	array[tid] = in[tid];

}

__device__ int &at(int *a, unsigned int i) {
	return a[i];
}

__device__ void insert_atomic(int *a, int e, int *size, int q) {
	int idx = atomicAdd(size, 1);
	a[idx] = e;
}

__global__ void test_insert(int *a, int *size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= *size) return;
	int idx = atomicAdd(size, 1);
	a[idx] = tid;
}


// tests
template <typename T>
__global__ void insert_template(T v, int n, int *size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= n) return;
	insert_atomic(v, at(v, tid), size, 1);
}

int main(int argc, char **argv){

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1000*NB*BSIZE*sizeof(int));
	hipDeviceSetLimit(hipLimitMallocHeapSize, INT_MAX*sizeof(int));

	int *a, *ha;
	int size = 1e5;
	int *static_size;
	int *memmap_size;
	ha = new int[size];
	for (int i = 0; i < size; ++i) {
		ha[i] = i;
	}
	gpuErrCheck( hipMalloc(&a, 2*size*sizeof(int)) );
	gpuErrCheck( hipMalloc(&static_size, sizeof(int)) );
	gpuErrCheck( hipMalloc(&memmap_size, sizeof(int)) );
	gpuErrCheck( hipMemcpy(a, ha, size*sizeof(int), hipMemcpyHostToDevice)) ;
	gpuErrCheck( hipMemcpy(static_size, &size, sizeof(int), hipMemcpyHostToDevice) );

	printf("%d\n", size);
	
	insert_template<int*><<<gridSize(size,BSIZE),  BSIZE>>>(a, size, static_size);
	
	gpuErrCheck( hipMemcpy(&size, static_size, sizeof(int), hipMemcpyDeviceToHost) );
	printf("%d\n", size);
	 
	// low level api
	hipSetDevice(0);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1e7*sizeof(int));
	hipCtx_t ctx;
	hipDevicePrimaryCtxRetain(&ctx, 0);
	hipCtxSetCurrent(ctx);

	VectorMemMap a2 = VectorMemMap(ctx);
	hipError_t status;
	gpuErrCheck( hipMemcpy(memmap_size, &size, sizeof(int), hipMemcpyHostToDevice) );

	status = a2.grow(4*size*sizeof(int));
	gpuErrCheck( hipMemcpy((void*)a2.getPointer(), ha, size*sizeof(int), hipMemcpyHostToDevice)) ;
	insert_template<hipDeviceptr_t><<<gridSize(size,BSIZE),BSIZE>>>(a2.getPointer(), size, memmap_size);
	gpuErrCheck( hipMemcpy(&size, memmap_size, sizeof(int), hipMemcpyDeviceToHost) );
	printf("%d\n", size);
	
	//size_t free;
	//hipMemGetInfo(&free, NULL);
	//cout << "Total Free Memory: " <<
		//(float)free << endl;
	
	//run_experiment(ctx);
	hipDevicePrimaryCtxRelease(0);
	
	return 0;
}
