#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "utility.cuh"

#define ull unsigned long long int
#define BSIZE 1024
//#define NB 64
#define PROB 90
#define FBS 1024
#define logFBS 10

inline __device__ int log2i32(unsigned int n){
	return __clz(n) ^ 31;
}

inline __device__ int log2i64(unsigned long long int n){
	return __clzll(n) ^ 63;
}

// LFVector
template <typename T>
struct LFVector {
	unsigned int size;
	T **a;
	int *isbucket;
	__device__ LFVector();
	__device__ T& at(unsigned int i);
	__device__ int get_bucket(unsigned int i);
	__device__ void new_bucket(unsigned int b);
	__device__ void push_back(T e);
	__device__ void grow(unsigned int n);
};

template <typename T>
__device__ LFVector<T>::LFVector() {
	size = 0;
	a = (T**)malloc(sizeof(T*)*64);
	a[0] = (T*)malloc(sizeof(T)*FBS);
	isbucket = (int*)malloc(sizeof(int)*64);
	isbucket[0] = 1;
	for (int i = 1; i < 64; ++i) {
		a[i] = nullptr;
		isbucket[i] = 0;
	}
}

template <typename T>
__device__ int LFVector<T>::get_bucket(unsigned int i) {
	return log2i32(i + FBS) - log2i32(FBS);
}

template <typename T>
__device__ T& LFVector<T>::at(unsigned int i) {
	int b = get_bucket(i);
	int pos = i + FBS;
	int idx = pos ^ (1 << log2i32(pos));
	return a[b][idx];
}

template <typename T>
__device__ void LFVector<T>::new_bucket(unsigned int b) {
	//printf("inside new_bucket %d\n", b);
	int old = atomicCAS(isbucket + b, 0, 1);
	if (old == 0) {
		int bsize = 1 << (logFBS + b);
		a[b] = (T*)malloc(sizeof(T) * bsize);
	}
	__syncthreads();
}

template <typename T>
__device__ void LFVector<T>::push_back(T e) {
	int idx = atomicAdd(&size, 1);
	int b = get_bucket(idx);
	while (a[b] == nullptr) {
		new_bucket(b);
	}
	at(idx) = e;
}

template <typename T>
__device__ void LFVector<T>::grow(unsigned int n) {
	int b1 = get_bucket(size);
	int b2 = get_bucket(n);
	for (int b = b1+1; b <= b2; ++b) {
		new_bucket(b);
		isbucket[b] = 1;
	}
}

template <typename T, int NB>
struct Vector {
	unsigned int size;
	LFVector<T> *lfv;
	unsigned int *ranges;
	__device__ Vector();
	__device__ T& at(unsigned int i);
	__device__ void insert(T e, int q);
	__device__ void grow(unsigned int n);
};

template <typename T, int NB>
__device__ Vector<T, NB>::Vector() {
	size = 0;
	lfv = (LFVector<T>*)malloc(sizeof(LFVector<T>)*NB);
	ranges = (unsigned int*)malloc(sizeof(unsigned int)*NB);
	for (int i = 0; i < NB; ++i) {
		ranges[i] = 0;
		lfv[i] = LFVector<T>();
	}
}

template <typename T, int NB>
__device__ T& Vector<T, NB>::at(unsigned int i) {
	// TODO use warp instructions
	int b = NB-1;
	while (i < ranges[b]) {
		--b;
	}
	return lfv[b].at(i-ranges[b]);
}

template <typename T, int NB>
__device__ void Vector<T, NB>::insert(T e, int q) {
	__shared__ int inserted;
	if (q == 0 && threadIdx.x != 0)
		return;
	if (threadIdx.x == 0)
		inserted = 0;
	__syncthreads();
	int bid = blockIdx.x;
	lfv[bid].push_back(e);
	atomicAdd(&inserted, 1);
	__syncthreads();
	if (threadIdx.x > blockIdx.x && threadIdx.x < NB)
		atomicAdd(ranges+threadIdx.x, inserted);
	//if (threadIdx.x == 0)
		//atomicAdd(&size, inserted);
		atomicAdd(&size, 1);
}

template <typename T, int NB>
__device__ void Vector<T, NB>:: grow(unsigned int n) {
	int tid = threadIdx.x;
	int sub_size = (n + NB - 1) / NB;
	lfv[tid].grow(sub_size);
}

template <typename T, int NB>
__global__ void growVec(Vector<T, NB> *v, unsigned int n) {
	v->grow(n);
}


template <typename T, int NB>
__global__ void createLFVector(Vector<T, NB> *v) {
	*v = Vector<T, NB>();
	return;
	v->size = 0;
	for (int i = 0; i < NB; ++i) {
		v->ranges[i] = 0;
		v->lfv[i].grow(0);
	}
}

template <typename T, int NB>
__global__ void initVec(Vector<T, NB> *v, unsigned int n) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bsize = n / NB;
	int start;
	if (bid < n % NB) {
		++bsize;
		start = bsize * bid;
	} else {
		start = bsize * bid + n%NB;
	}
	for (int i = tid; i < bsize && bid*NB+i < n; i += BSIZE) {
		v->insert(start+i, 1);
	}
}

template <typename T, int NB>
__global__ void initVec(Vector<T, NB> *v, unsigned int n, T* in) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bsize = n / NB;
	int start;
	if (bid < n % NB) {
		++bsize;
		start = bsize * bid;
	} else {
		start = bsize * bid + n%NB;
	}
	for (int i = tid; i < bsize && bid*NB+i < n; i += BSIZE) {
		v->insert(in[start+i], 1);
	}
}

template<typename T, int NB>
__global__ void get_size(int *out, Vector<T, NB> *v) {
	*out = v->size;
}

template<typename T, int NB>
__global__ void vec2array(T *out, Vector<T, NB> *v) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	out[tid] = v->at(tid);
}

template<typename T, int NB>
int sendToHost(T* &out, Vector<T, NB> *v) {
	int *ds, size, *temp;
	gpuErrCheck( hipMalloc(&ds, sizeof(int)) );
	get_size<<<1,1>>>(ds, v);
	gpuErrCheck( hipMemcpy(&size, ds, sizeof(int), hipMemcpyDeviceToHost) );

	out = (T*)malloc(size * sizeof(T));
	gpuErrCheck( hipMalloc(&temp, size*sizeof(T)) );
	vec2array<<<gridSize(size, BSIZE), BSIZE>>>(temp, v);
	gpuErrCheck( hipMemcpy(out, temp, size*sizeof(int), hipMemcpyDeviceToHost) );
	return size;
}

// LFVector test
template<int NB>
__global__ void printVec(Vector<int, NB> *v) {
	printf("size: %d\n", v->size);
	// pint lfv[0]
	for (int i = 0; i < 64; ++i) {
		printf("%d ", v->lfv[0].isbucket[i]);
	}
	
	return;
	printf("ranges: ");
	for (int i = 0; i < NB; ++i) {
		printf("%d ", v->ranges[i]);
	}
	printf("\nsizes: ");
	for (int i = 0; i < NB; ++i) {
		printf("%d ", v->lfv[i].size);
	}
	printf("\n");
	return;
	for (int i = 0; i < v->size; ++i) {
		printf("%d ", v->at(i));
	}
	printf("\n");
	printf("last element: %d\n", v->at(v->size - 1));
}

template<int NB>
__global__ void initVec(Vector<int, NB> *v) {
	*v = Vector<int, NB>();
	v->size = NB*BSIZE;
	for (int i = 0; i < NB; ++i) {
		v->ranges[i] = BSIZE*i;
		v->lfv[i].grow(BSIZE);
	}
	for(int i = 0; i < v->size; ++i) {
		v->at(i) = i;
	}
}

template<int NB>
__global__ void test_insert(Vector<int, NB> *v) {
	int tid = threadIdx.x;
	v->insert(tid, 1);
}

template<int NB>
__global__ void test_insert2(Vector<int, NB> *v) {
	int tid = threadIdx.x;
	int bs = v->lfv[blockIdx.x].size;
	//printf("%d %d %d\n", tid, blockIdx.x, bs);
	for (int i = tid; i < bs; i += BSIZE) {
		v->insert(i, 1);
	}
}

template<int NB>
__global__ void test_insert2_2(Vector<int, NB> *v) {
	int tid = threadIdx.x;
	int bs = v->lfv[blockIdx.x].size;
	//printf("%d %d %d\n", tid, blockIdx.x, bs);
	for (int i = tid; i < bs; i += BSIZE) {
		v->insert(v->lfv[blockIdx.x].at(i)+1024, 1);
	}
}

template<int NB>
__global__ void test_insert3(Vector<int, NB> *v) {
	int tid = threadIdx.x;
	for (int i = 0; i < 10; ++i) {
		v->insert(tid, 1);
	}
}

template<int NB>
__global__ void test_read_write_g(Vector<int, NB> *v, int size, int rep) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= size) return;
	for (int i = 0; i < rep; ++i) {
		v->at(tid) += 1;
	}
}

template<int NB>
__global__ void test_read_write_b(Vector<int, NB> *v, int rep) {
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	for (int i = tid; i < v->lfv[bid].size; i += BSIZE) {
		for (int j = 0; j < rep; ++j) {
			v->lfv[bid].at(i) += 1;
		}
	}
}

__global__ void random_copy(LFVector<int> *v) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int n = v->size;
	for (int i = tid; i < n; i+=BSIZE) {
		if (v->at(i) < PROB)
			v->push_back(v->at(i));
	}
}

__global__ void init_random(LFVector<int> *v, int *A, int n) {
	v->size = 0;
	v->a = (int**)malloc(sizeof(int*)*100);
	v->a[0] = (int*)malloc(sizeof(int)*32);
	v->isbucket = (int*)malloc(sizeof(int)*100);
	v->isbucket[0] = 1;
	for (int i = 1; i < 100; ++i) {
		v->a[i] = nullptr;
		v->isbucket[i] = 0;
	}
	v->grow(n);
	for (int i = 0; i < n; ++i) {
		v->at(i) = A[i];
	}
}

void test_random_copy(LFVector<int> *v, int n) {
	int *hA, *dA;
	hA = new int[n];
	for (int i = 0; i < n; ++i) {
		hA[i] = rand() % 101;
	}

	hipMalloc(&dA, sizeof(int)*n);
	hipMemcpy(dA, hA, sizeof(int)*n, hipMemcpyHostToDevice);
	init_random<<<1,1>>>(v, dA, n);
	//printLFVec<<<1,1>>>(v); kernelCallCheck();

	hipEvent_t start, stop;
	start_clock(start, stop);

	//random_copy<<<gridSize(n, BSIZE), BSIZE>>>(v);
	random_copy<<<1, BSIZE>>>(v);
	kernelCallCheck();

	float time =stop_clock(start, stop);
	//printLFVec<<<1,1>>>(v); kernelCallCheck();
	//printf("time: %f ms\n", time);
	printf("%f\n", time);
}

template<int NB>
void run_experiment(Vector<int, NB> *v, int size, int ratio) {
	int *ds;
	gpuErrCheck( hipMalloc(&ds, sizeof(int)) );

	int rep = 10;
	int size_exp = 29 - rep;
	size = 1 << size_exp;
	int rw_rep = 30;
	int o_size = size;
	createLFVector<<<1,1>>>(v); kernelCallCheck();
	initVec<<<NB,BSIZE>>>(v, size); kernelCallCheck();
	//printVec<<<1,1>>>(v); kernelCallCheck();
	float results[rep];
	float results_grow[rep];
	float results_rw[rw_rep];
	int rw_kernel_rep = 1;

	for (int i = 0; i < rep; ++i) {
		printf("%d ", i); fflush(stdout);

		// grow
		hipEvent_t start, stop;
		start_clock(start, stop);
		growVec<<<1,NB>>>(v, 2*size);
		hipDeviceSynchronize();
		results_grow[i] = stop_clock(start, stop);
		//printVec<<<1,1>>>(v); kernelCallCheck();

		// insertion
		start_clock(start, stop);
		test_insert2<<<NB, BSIZE>>>(v); kernelCallCheck();
		hipDeviceSynchronize();
		results[i] = stop_clock(start, stop);

		// read/write
		results_rw[i] = 0.0;
		for (int j = 0; j < rw_rep; ++j) {
			hipEvent_t start, stop;
				get_size<<<1,1>>>(ds, v);
				hipMemcpy(&size, ds, sizeof(int), hipMemcpyDeviceToHost);
			start_clock(start, stop);
			// wr block
				test_read_write_b<<<NB, BSIZE>>>(v, rw_kernel_rep);
			// wr global - slow
				//test_read_write_g<<<gridSize(size, BSIZE), BSIZE>>>(v, size, rw_kernel_rep);
			hipDeviceSynchronize();
			results_rw[i] += stop_clock(start, stop);
		}
		results_rw[i] /= rw_rep;
		size *= 2;
	}
	
	// print results
	printf("\n");
	// grow
	printf("mlfv%d,grow,%d,%d,", NB, o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results_grow[i]);
	}
	printf("%f\n", results_grow[rep-1]);
	// insert
	printf("mlfv%d,in,%d,%d,", NB, o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results[i]);
	}
	printf("%f\n", results[rep-1]);
	//printf("%f\n", s);
	// read-write
	printf("mlfv%d,rw%d,%d,%d,", NB, rw_kernel_rep, o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results_rw[i]);
	}
	printf("%f\n", results_rw[rep-1]);
}



int main(int argc, char **argv){

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1000*NB*BSIZE*sizeof(int));
	hipDeviceSetLimit(hipLimitMallocHeapSize, INT_MAX*sizeof(int));

	int *a, *ha;
	int size = 1<<19;
	//int size = 1e6;
	ha = new int[size];
	for (int i = 0; i < size; ++i) {
		ha[i] = i;
	}
	gpuErrCheck( hipMalloc(&a, size*sizeof(int)) );
	gpuErrCheck( hipMemcpy(a, ha, size*sizeof(int), hipMemcpyHostToDevice)) ;
	
	// LFV
	const int NB = 64;
	Vector<int, NB> *lfv;
	gpuErrCheck( hipMalloc(&lfv, sizeof(Vector<int, NB>)) );

	//createLFVector<<<1,1>>>(lfv); kernelCallCheck();
	//initVec<<<NB,BSIZE>>>(lfv, size, a); kernelCallCheck();
	//printVec<<<1,1>>>(lfv); kernelCallCheck();

	// TODO use ratio (3rd arg) in insertion
	run_experiment<NB>(lfv, size, 1);

	return 0;

	//initVec<<<1,1>>>(lfv); kernelCallCheck();
	createLFVector<<<1,1>>>(lfv); kernelCallCheck();
	initVec<<<NB,BSIZE>>>(lfv, size, a); kernelCallCheck();
	//printVec<<<1,1>>>(lfv); kernelCallCheck();
	int *r;
	int final_size = sendToHost(r, lfv);
	printf("%d\n", final_size);
	//print_array(r, final_size);
	for (int i = 0; i < 5; ++i) {
		printf("%d\n", i);
		test_insert2_2<<<NB,BSIZE>>>(lfv); kernelCallCheck();
		printVec<<<1,1>>>(lfv); kernelCallCheck();
	}
	//printVec<<<1,1>>>(lfv); kernelCallCheck();
	
	return 0;
}
