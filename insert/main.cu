#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <mma.h>
#include "../common/utility.cuh"

#define ull unsigned long long int
#define WARPSIZE 32
#define BSIZE 1024
#define NB 100
#define PROB 90
#define DEBUG 1

using namespace nvcuda;


// atomic
__global__ void atomic(int *C, int *A, int *s, int n) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n) return;
        C[tid] = atomicAdd(s, A[tid]);
}

__device__ void insert_atomic1(int *a, int e, int *size, int q) {
	__shared__ int ss;
	__shared__ int idx_g;
	int idx_b;
	if (threadIdx.x == 0)
		ss = 0;
	__syncthreads();
	if (q) {
		int idx_b = atomicAdd(size, 1);
		//a[idx] = e;
	}
	__syncthreads();
	if (threadIdx.x == 0)
		idx_g = atomicAdd(size, ss);
	__syncthreads();
	if (q)
		a[idx_g + idx_b] = e;
}
__device__ void insert_atomic(int *a, int e, int *size, int q) {
	if (threadIdx.x == 0)
		printf("bid %i, s %i\n", blockIdx.x, *size);
	if (q) {
		int idx = atomicAdd(size, 1);
		a[idx] = e;
	}
}


// scan
__inline__ __device__ int warp_scan(int val, int lane){
	for (int offset = 1; offset < WARPSIZE; offset <<= 1) {
                int n = __shfl_up_sync(0xffffffff, val, offset, WARPSIZE);
		if ((lane & 31) >= offset)
			val += n;
	}
	return val;
}

__inline__ __device__ int block_scan(int val){
        static __shared__ int shared[WARPSIZE];
        int tid = threadIdx.x;
        int lane = tid & (WARPSIZE-1);
        int wid = tid/WARPSIZE;
        val = warp_scan(val, lane);
        if(lane == WARPSIZE-1)
                shared[wid] = val;

        __syncthreads();
        if(wid == 0){
                int t = (tid < blockDim.x/WARPSIZE) ? shared[lane] : 0;
                t = warp_scan(t, lane);
                shared[lane] = t;
        }
        __syncthreads();
        if (wid > 0){
                val += shared[wid-1];
        }
        return val;
}

__global__ void scan(int *C, int *A, int *s, int n) {
        __shared__ int ss;
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n) return;
        int val = block_scan(A[tid]);
        if (threadIdx.x == BSIZE - 1 || tid == n - 1) {
                ss = atomicAdd(s, val);
        }
        __syncthreads();
        C[tid] = val + ss - A[tid];
}

__device__ void insert_scan(int *a, int e, int *size, int q) {
        __shared__ int ss;
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int n = *size;
	if (tid >= n) return;
        int val = block_scan(q);
        if (threadIdx.x == BSIZE - 1 || tid == n - 1) {
                ss = atomicAdd(size, val);
        }
        __syncthreads();
        int idx = val + ss - q;
	//printf("tid %d: %d", tid, idx);
	if (q)
		a[idx] = e;
}


// tensor core scan
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

/*
static __device__ half upper_triang[256];
static __device__ half lower_triang[256];
void load_triang_matrices() {
	half upper[256];
	half lower[256];
	for (int i = 0; i < 16; ++i) {
		for (int j = 0; j < 16; ++j) {
			int tid = i*16 + j;
			if (i <= j) {
				upper[tid] = 1.0;
				lower[tid] = 0.0;
			} else {
				upper[tid] = 0.0;
				lower[tid] = 1.0;
			}
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(upper_triang), upper, sizeof(upper)); kernelCallCheck();
	hipMemcpyToSymbol(HIP_SYMBOL(lower_triang), lower, sizeof(lower)); kernelCallCheck();
}
__global__ void load_matrices() {
	int tid = threadIdx.x;
	int i = tid / 16;
	int j = tid % 16;
	if (i <= j) {
		upper_triang[tid] = 1.0;
		lower_triang[tid] = 0.0;
	} else {
		upper_triang[tid] = 0.0;
		lower_triang[tid] = 1.0;
	}
}
*/

__device__ float tensor_block_scan(half val,
		half vals[BSIZE],
		half upper_triang[256],
		half lower_triang[256],
		half add[256]) {
	int tid = threadIdx.x;
	if  (tid < 256) {
		int i = tid / 16;
		int j = tid % 16;
		if (i <= j) {
			upper_triang[tid] = 1;
			lower_triang[tid] = 0;
		} else {
			upper_triang[tid] = 0;
			lower_triang[tid] = 1;
		}
	}
	int wid = tid / WARPSIZE;
	vals[tid] = val;
	__syncthreads();

	if (wid < 4) {
		wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
		wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
		wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;
		wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> au_frag;
		wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> la_frag;

		// AU = A * UPPER + 0
		wmma::fill_fragment(c_frag, 0.0f);
		wmma::load_matrix_sync(a_frag, vals + 256*wid, 16);
		wmma::load_matrix_sync(b_frag, upper_triang, 16);

		wmma::mma_sync(au_frag, a_frag, b_frag, c_frag);

		// LA = LOWER * A + 0
		wmma::load_matrix_sync(a_frag, lower_triang, 16);
		wmma::load_matrix_sync(b_frag, vals, 16);

		wmma::mma_sync(la_frag, a_frag, b_frag, c_frag);

		// R = LA * 1 + AU
		wmma::store_matrix_sync(vals + wid*256, la_frag, 16,  wmma::mem_row_major);
		wmma::load_matrix_sync(a_frag, vals + wid*256, 16);
		wmma::fill_fragment(b_frag, 1.0f);

		wmma::mma_sync(c_frag, a_frag, b_frag, au_frag);

		wmma::store_matrix_sync(vals + wid*256, c_frag, 16,  wmma::mem_row_major);
	}
	__syncthreads();

	// combine warps
	if (tid == 0) {
		half i = 0;
		add[0] = 0;
		for (int j = 1; j < 4; ++j) {
			i += vals[j*256 - 1];
			add[j] = i;
		}
	}
	__syncthreads();
	vals[tid] += add[tid / 256];
	return vals[tid];
}

__global__ void tensor_scan(int *C, int *A, int *s, int n) {
	__shared__ half vals[BSIZE];
	__shared__ half upper[256];
	__shared__ half lower[256];
	__shared__ half add[4];
	__shared__ int ss;
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float pval = tid < n ? A[tid] : 0;
	//printf("n  tid %d: %f\n", tid, pval);
	__syncthreads();
	int val = tensor_block_scan(pval, vals, upper, lower, add);
	__syncthreads();
	printf("tid %d: %d\n", tid, val);
	if (tid < n) {
		if (threadIdx.x == BSIZE - 1 || tid == n - 1) {
			ss = atomicAdd(s, val);
		}
		__syncthreads();
		C[tid] = val + ss - A[tid];
	}
}

__global__ void test_block(int *out) {
	int tid = threadIdx.x;
	//int val = tensor_block_scan(1);
	//out[tid] = (int)val;
}

__device__ void insert_tensor_scan(int *a, int e, int *size, int q) {
	__shared__ half vals[BSIZE];
	__shared__ half upper[256];
	__shared__ half lower[256];
	__shared__ half add[4];
	__shared__ int ss;
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int n = *size;
	float pval = tid < n ? q : 0;
	//printf("n  tid %d: %f\n", tid, pval);
	__syncthreads();
	int val = tensor_block_scan(pval, vals, upper, lower, add);
	__syncthreads();
	//printf("tid %d: %d\n", tid, val);
	if (tid < n) {
		if (threadIdx.x == BSIZE - 1 || tid == n - 1) {
			ss = atomicAdd(size, val);
			//printf("bid %d: idx0: %d, size: %d\n", blockIdx.x, a[n+blockIdx.x], *size);
		}
		__syncthreads();
		int idx = val + ss - q;
		//printf("idx tid %d: %d\n", tid, idx);
		if (q)
			a[idx] = e;
	}
}


// test
__global__ void test_insert_atomic(int* v, int n, int *size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int q = tid >= n ? 0 : 1;
	if (tid > n) return;
	insert_atomic(v, tid, size, q);
	//if (tid == n-1)
		//printf("tid %i  *size %i\n", tid, *size);
}

__global__ void test_insert_scan(int* v, int n, int *size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int q = tid >= n ? 0 : 1;
	insert_scan(v, tid, size, q);
}

__global__ void test_insert_tensor_scan(int* v, int n, int *size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int q = tid >= n ? 0 : 1;
	insert_tensor_scan(v, tid, size, q);
}

void test_scan(int n) {
	int *hA, *dA, *hC, *dC, *ds, hs;
        hA = new int[n];
	hC = new int[n];
        for (int i = 0; i < n; ++i) {
                hA[i] = 1;
                //hA[i] = i%2;
        }
        hs = 0;
        hipMalloc(&dA, sizeof(int)*n);
        hipMalloc(&dC, sizeof(int)*n);
        hipMalloc(&ds, sizeof(int));
        hipMemcpy(dA, hA, sizeof(int)*n, hipMemcpyHostToDevice);
        hipMemcpy(ds, &hs, sizeof(int), hipMemcpyHostToDevice);

	//test_block<<<1,1024>>>(dC); kernelCallCheck();
	//hipMemcpy(hC, dC, 1024*sizeof(int), hipMemcpyDeviceToHost);
	//print_array(hC, 1024);


	//return 0;
        hipEvent_t start1, stop1;
        start_clock(start1, stop1);
	{
		//atomic<<<gridSize(n, BSIZE), BSIZE>>>(dC, dA, ds, n); kernelCallCheck();
		//scan<<<gridSize(n, BSIZE), BSIZE>>>(dC, dA, ds, n); //kernelCallCheck();
		tensor_scan<<<gridSize(n, BSIZE), BSIZE>>>(dC, dA, ds, n); kernelCallCheck();
	}
        float time = stop_clock(start1, stop1);

	if (DEBUG) {
		gpuErrCheck( hipMemcpy(hC, dC, n*sizeof(int), hipMemcpyDeviceToHost) );
		print_array(hA, n, "array A:");
		print_array(hC, n, "array C:");
	}


        printf("time: %f ms\n", time);
        printf("%f,", time);

}

void test_insert_scan(int size) {
	int *a, *ha;
	int *dsize;
	ha = new int[size];
	for (int i = 0; i < size; ++i) {
		ha[i] = i;
	}
	gpuErrCheck( hipMalloc(&a, 2*size*sizeof(int)) );
	gpuErrCheck( hipMalloc(&dsize, sizeof(int)) );
	gpuErrCheck( hipMemcpy(a, ha, size*sizeof(int), hipMemcpyHostToDevice)) ;
	gpuErrCheck( hipMemcpy(dsize, &size, sizeof(int), hipMemcpyHostToDevice) );

	test_insert_scan<<<gridSize(size, BSIZE), BSIZE>>>(a, size, dsize); kernelCallCheck();
	hipDeviceSynchronize();
	
	int *ha2 = new int[2*size];
	hipMemcpy(ha2, a, sizeof(int)*size*2, hipMemcpyDeviceToHost);
	print_array(ha2, size*2);

}

void run_experiment(int insert_function) {
	int size = 1<<19;
	int ratio = 1;
	int rep = 10;
	int o_size = size;

	int *a, *ha;
	int *dsize;
	ha = new int[size];
	for (int i = 0; i < size; ++i) {
		ha[i] = i;
	}
	gpuErrCheck( hipMalloc(&a, 2*size*2^rep*sizeof(int)) );
	gpuErrCheck( hipMalloc(&dsize, sizeof(int)) );
	gpuErrCheck( hipMemcpy(a, ha, size*sizeof(int), hipMemcpyHostToDevice)) ;
	gpuErrCheck( hipMemcpy(dsize, &size, sizeof(int), hipMemcpyHostToDevice) );
	gpuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_size), &size, sizeof(int)) );


	float results[rep];

	for (int i = 0; i < rep; ++i) {
		fprintf(stderr, "%d %d \n", i, size);
		hipEvent_t start, stop;
		start_clock(start, stop);
		switch (insert_function) {
			case 0: test_insert_atomic<<<gridSize(size, BSIZE), BSIZE>>>(a, size, dsize);
				break;
			case 1: test_insert_scan<<<gridSize(size, BSIZE), BSIZE>>>(a, size, dsize);
				break;
			case 2: test_insert_tensor_scan<<<gridSize(size, BSIZE), BSIZE>>>(a, size, dsize);
				break;
		}
		kernelCallCheck();
		hipDeviceSynchronize();
		results[i] = stop_clock(start, stop);
		//hipMemcpy(&size, dsize, sizeof(int), hipMemcpyDeviceToHost);
		//gpuErrCheck( hipMemcpyFromSymbol(&size, HIP_SYMBOL(d_size), sizeof(int)) );
		size = size * 2;
	}

	// print results
	printf("static,in,%d,%d,", o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results[i]);
	}
	printf("%f\n", results[rep-1]);
	//printf("%f\n", s);
}


int main(int argc, char **argv){
	if (argc < 2) {
                fprintf(stderr, "Ejecutar como ./prog insert_fun\n");
                return -1;
        }
        //int size = atoi(argv[1]);
        int mode = atoi(argv[1]);

	//load_matrices<<<1,256>>>(); kernelCallCheck();

	//if (mode == 0)
		//test_scan(size);
	//else if (mode == 1)
		//test_insert_scan(size);
	run_experiment(mode);


	return 0;
}
