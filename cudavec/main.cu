#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include "utility.cuh"
using namespace std;


class VectorMemMap {
private:
    hipDeviceptr_t d_p;
    hipMemAllocationProp prop;
    hipMemAccessDesc accessDesc;
    struct Range {
        hipDeviceptr_t start;
        size_t sz;
    };
    std::vector<Range> va_ranges;
    std::vector<hipMemGenericAllocationHandle_t> handles;
    std::vector<size_t> handle_sizes;
    size_t alloc_sz;
    size_t reserve_sz;
    size_t chunk_sz;
public:
    VectorMemMap(hipCtx_t context);
    ~VectorMemMap();

    hipDeviceptr_t getPointer() const {
        return d_p;
    }

    size_t getSize() const {
        return alloc_sz;
    }
    size_t getReserve() const {
        return reserve_sz;
    }

    // Reserves some extra space in order to speed up grow()
    hipError_t reserve(size_t new_sz);

    // Actually commits num bytes of additional memory
    hipError_t grow(size_t new_sz);
};

VectorMemMap::VectorMemMap(hipCtx_t context) : d_p(0ULL), prop(), handles(), alloc_sz(0ULL), reserve_sz(0ULL), chunk_sz(0ULL)
{
    hipDevice_t device;
    hipCtx_t prev_ctx;
    hipError_t status = hipSuccess;
    (void)status;

    status = hipCtxGetCurrent(&prev_ctx);
    assert(status == hipSuccess);
    if (hipCtxSetCurrent(context) == hipSuccess) {
        status = hipCtxGetDevice(&device);
        assert(status == hipSuccess);
        status = hipCtxSetCurrent(prev_ctx);
        assert(status == hipSuccess);
    }

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = (int)device;
    prop.win32HandleMetaData = NULL;

    accessDesc.location = prop.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    status = hipMemGetAllocationGranularity(&chunk_sz, &prop, hipMemAllocationGranularityMinimum);
    assert(status == hipSuccess);
}

VectorMemMap::~VectorMemMap()
{
    hipError_t status = hipSuccess;
    (void)status;
    if (d_p != 0ULL) {
        status = hipMemUnmap(d_p, alloc_sz);
        assert(status == hipSuccess);
        for (size_t i = 0ULL; i < va_ranges.size(); i++) {
            status = hipMemAddressFree(va_ranges[i].start, va_ranges[i].sz);
            assert(status == hipSuccess);
        }
        for (size_t i = 0ULL; i < handles.size(); i++) {
            status = hipMemRelease(handles[i]);
            assert(status == hipSuccess);
        }
    }
}

hipError_t
VectorMemMap::reserve(size_t new_sz)
{
    hipError_t status = hipSuccess;
    hipDeviceptr_t new_ptr = 0ULL;

    if (new_sz <= reserve_sz) {
        return hipSuccess;
    }

    const size_t aligned_sz = ((new_sz + chunk_sz - 1) / chunk_sz) * chunk_sz;

    status = hipMemAddressReserve(&new_ptr, (aligned_sz - reserve_sz), 0ULL, d_p + reserve_sz, 0ULL);

    // Try to reserve an address just after what we already have reserved
    if (status != hipSuccess || (new_ptr != d_p + reserve_sz)) {
        if (new_ptr != 0ULL) {
            (void)hipMemAddressFree(new_ptr, (aligned_sz - reserve_sz));
        }
        // Slow path - try to find a new address reservation big enough for us
        status = hipMemAddressReserve(&new_ptr, aligned_sz, 0ULL, 0U, 0);
        if (status == hipSuccess && d_p != 0ULL) {
            hipDeviceptr_t ptr = new_ptr;
            // Found one, now unmap our previous allocations
            status = hipMemUnmap(d_p, alloc_sz);
            assert(status == hipSuccess);
            for (size_t i = 0ULL; i < handles.size(); i++) {
                const size_t hdl_sz = handle_sizes[i];
                // And remap them, enabling their access
                if ((status = hipMemMap(ptr, hdl_sz, 0ULL, handles[i], 0ULL)) != hipSuccess)
                    break;
                if ((status = hipMemSetAccess(ptr, hdl_sz, &accessDesc, 1ULL)) != hipSuccess)
                    break;
                ptr += hdl_sz;
            }
            if (status != hipSuccess) {
                // Failed the mapping somehow... clean up!
                status = hipMemUnmap(new_ptr, aligned_sz);
                assert(status == hipSuccess);
                status = hipMemAddressFree(new_ptr, aligned_sz);
                assert(status == hipSuccess);
            }
            else {
                // Clean up our old VA reservations!
                for (size_t i = 0ULL; i < va_ranges.size(); i++) {
                    (void)hipMemAddressFree(va_ranges[i].start, va_ranges[i].sz);
                }
                va_ranges.clear();
            }
        }
        // Assuming everything went well, update everything
        if (status == hipSuccess) {
            Range r;
            d_p = new_ptr;
            reserve_sz = aligned_sz;
            r.start = new_ptr;
            r.sz = aligned_sz;
            va_ranges.push_back(r);
        }
    }
    else {
        Range r;
        r.start = new_ptr;
        r.sz = aligned_sz - reserve_sz;
        va_ranges.push_back(r);
        if (d_p == 0ULL) {
            d_p = new_ptr;
        }
        reserve_sz = aligned_sz;
    }

    return status;
}

hipError_t
VectorMemMap::grow(size_t new_sz)
{
    hipError_t status = hipSuccess;
    hipMemGenericAllocationHandle_t handle;
    if (new_sz <= alloc_sz) {
        return hipSuccess;
    }

    const size_t size_diff = new_sz - alloc_sz;
    // Round up to the next chunk size
    const size_t sz = ((size_diff + chunk_sz - 1) / chunk_sz) * chunk_sz;

    if ((status = reserve(alloc_sz + sz)) != hipSuccess) {
        return status;
    }

    if ((status = hipMemCreate(&handle, sz, &prop, 0)) == hipSuccess) {
        if ((status = hipMemMap(d_p + alloc_sz, sz, 0ULL, handle, 0ULL)) == hipSuccess) {
            if ((status = hipMemSetAccess(d_p + alloc_sz, sz, &accessDesc, 1ULL)) == hipSuccess) {
                handles.push_back(handle);
                handle_sizes.push_back(sz);
                alloc_sz += sz;
            }
            if (status != hipSuccess) {
                (void)hipMemUnmap(d_p + alloc_sz, sz);
            }
        }
        if (status != hipSuccess) {
            (void)hipMemRelease(handle);
        }
    }

    return status;
}

__device__ int &at(hipDeviceptr_t d_p, unsigned int i) {
	return (*(int**)&d_p)[i];
}

__device__ void insert_atomic(hipDeviceptr_t d_p, int e, int *size, int q) {
	int idx = atomicAdd(size, 1);
	at(d_p, idx) = e;
}

__global__ void initVec(hipDeviceptr_t d_p, unsigned int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n) return;
	at(d_p, tid) = tid;
}

__global__ void initVec(hipDeviceptr_t d_p, unsigned int n, int* in) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n) return;
	at(d_p, tid) = in[tid];
}

__global__ void test(hipDeviceptr_t d_p, size_t n) {
	for (size_t i = 0; i < n; ++i) {
		at(d_p, i) = i;
	}
}

__global__ void printVec(hipDeviceptr_t d_p, size_t n) {
	for (size_t i = 0; i < n; ++i) {
		printf("%d ", at(d_p, i));
	}
	printf("\n");
}

__global__ void test_insert_atomic(hipDeviceptr_t v, int n, int *size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= n) return;
	insert_atomic(v, at(v, tid), size, 1);
}

__global__ void test_read_write(hipDeviceptr_t v, int size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= size) return;
	at(v, tid) += 1;
}

// low level api test
void run_experiment(hipCtx_t ctx, int size, int ratio) {
	int rep = 10;
	int rw_rep = 30;
	int o_size = size;
	int *ds;
	hipMalloc(&ds, sizeof(int));
	hipMemcpy(ds, &size, sizeof(int), hipMemcpyHostToDevice);

	VectorMemMap a = VectorMemMap(ctx);
	hipError_t status;

	status = a.grow(size*sizeof(int));
	initVec<<<gridSize(size, 1024), 1024>>>(a.getPointer(), size); kernelCallCheck();

	float results[rep];
	float results_rw[rw_rep];
	
	for (int i = 0; i < rep; ++i) {
		hipEvent_t start, stop;
		start_clock(start, stop);
		status = a.grow(size*2*sizeof(int));
		test_insert_atomic<<<gridSize(size, 1024), 1024>>>(a.getPointer(), size, ds);
		results[i] = stop_clock(start, stop);
		hipMemcpy(&size, ds, sizeof(int), hipMemcpyDeviceToHost);

		// read/write
		results_rw[i] = 0.0;
		for (int j = 0; j < rw_rep; ++j) {
			hipEvent_t start, stop;
			start_clock(start, stop);
			test_read_write<<<gridSize(size, 1024), 1024>>>(a.getPointer(), size); kernelCallCheck();
			results_rw[i] += stop_clock(start, stop);
		}
		results_rw[i] /= rw_rep;
	}

	// print results
	printf("memMap,%d,%d,", o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results[i]);
	}
	printf("%f\n", results[rep-1]);
	//printf("%f\n", s);
	printf("memMap,%d,%d,", o_size, ratio);
	for (int i = 0; i < rep-1; ++i) {
		printf("%f,", results_rw[i]);
	}
	printf("%f\n", results_rw[rep-1]);
}

int main(int argc, char **argv){
	int size = 1e6;
	int ratio = 2;
	hipSetDevice(0);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1e7*sizeof(int));
	hipCtx_t ctx;
	hipDevicePrimaryCtxRetain(&ctx, 0);
	hipCtxSetCurrent(ctx);
	
	//size_t free;
	//hipMemGetInfo(&free, NULL);
	//cout << "Total Free Memory: " <<
		//(float)free << endl;
	
	run_experiment(ctx, size, ratio);

	/*
	VectorMemMap a = VectorMemMap(ctx);
	hipError_t status;

	cout << "size: " << a.getSize() << endl;
	status = a.reserve(n*sizeof(int));
	cout << "size: " << a.getSize() << endl;
	status = a.grow(n*sizeof(int));
	cout << "size: " << a.getSize() << " reserve: " << a.getReserve() << endl;

	test<<<1,1>>>(a.getPointer(), n); kernelCallCheck();
	//printVec<<<1,1>>>(a.getPointer(), n); kernelCallCheck();
	
	//status = a.grow(1000000);
	//cout << "size: " << a.getSize() << " reserve: " << a.getReserve() << endl;

	n = 1e7;
	status = a.grow(n*sizeof(int));
	cout << "status: " << status << endl;
	cout << "size: " << a.getSize() << " reserve: " << a.getReserve() << endl;
	test<<<1,1>>>(a.getPointer(), n); kernelCallCheck();
	*/
	
	hipDevicePrimaryCtxRelease(0);
	return 0;
}

